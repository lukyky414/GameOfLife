#include "hip/hip_runtime.h"
#include "calcul.cuh"

//Calcul d'une génération
__global__ void data_cuda(unsigned char* in, unsigned char* out, unsigned long rule){
    uint x = threadIdx.x + blockIdx.x * NB_THREAD;
    if(x >= TEXTUR_COL)
        return;

    int i;
    //L'état d'une cellule est défini par son voisinage
    unsigned long state = 0;

    for(i=-VOISINAGE; i <= VOISINAGE; i++){
        state = state << 1;
        state += in[(x+TEXTUR_COL+i)%TEXTUR_COL];
    }

    out[x] = (rule&(1l<<state))>>state;
}

//Calcul de la texture
__global__ void texture_cuda(unsigned char* data, uchar4* texture, uint y){
    uint x = threadIdx.x + blockIdx.x*NB_THREAD;
    if(x >= TEXTUR_COL)
        return;
    uint pos = x + y*TEXTUR_COL;

    if(data[x]){
        texture[pos].x = 255;
        texture[pos].y = 255;
        texture[pos].z = 255;
    }
    else{
        texture[pos].x = 0;
        texture[pos].y = 0;
        texture[pos].z = 0;
    }
}

extern unsigned char* host_data;
extern unsigned char* data1;
//Générer un ruban aléatoire
void random_data(){
    uint i;

    for(i=0; i<TEXTUR_COL; i++)
        host_data[i] = rand()%2;

    hipMemcpy(data1, host_data, TEXTUR_COL, hipMemcpyHostToDevice); hipDeviceSynchronize();
}

//Générer un ruban avec une seule cellule active
void initial_data(){
    uint i;

    for(i=0; i<TEXTUR_COL; i++)
        host_data[i] = 0;
    
    host_data[TEXTUR_COL/2] = 1;

    hipMemcpy(data1, host_data, TEXTUR_COL, hipMemcpyHostToDevice); hipDeviceSynchronize();
}