
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

//Nombre de block device
#define BLOCK 4096
//Nombre de thread device
#define THREAD 4096

//les fonctions avec __global__ seront execute sur la CG (device)
__global__ void mult(int* a, int* b, int* c){
    //Tableau des index pour deux blocks et trois threads:
    // [t1,t2,t3][t1,t2,t3]
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    c[index] = a[index] * b[index];
}

void random_ints(int* tab, int n){
    int i;
    srand (time (NULL));

    for(i=0; i<n; i++)
        tab[i] = rand();
}

//Le reste est compile avec le compilateur de base genre gcc
int main(void) {
    int N = BLOCK * THREAD;
    int size = N * sizeof(int);
    int i;
    clock_t time1, time2;

    //Variables présente sur le processeur (host)
    int *h_a, *h_b, *h_c;

    //Pointeurs pour le device memory
    int *d_a, *d_b, *d_c;

    
    //Alloue la mémoire device
    printf("Allocation Device\n");
    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    //Alloue la mémoire host
    printf("Allocation Host\n");
    h_a = (int*) malloc (size); random_ints(h_a,N);
    h_b = (int*) malloc (size); random_ints(h_b,N);
    h_c = (int*) malloc (size);

    //Copie les valeurs dans la device memory
    printf("Copie:\n");
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    //Calcul de multiplication en parallèle
    printf("Calcul en parallele:");
    time1 = clock();
    mult<<<BLOCK,THREAD>>>(d_a, d_b, d_c);
    time2 = clock();
    printf("%d\n", time2-time1);

    printf("Calcul en local:");
    time1 = clock();
    for(i=0; i<N; i++)
        h_c[i] = h_a[i] * h_b[i];
    time2 = clock();
    printf("%d\n", time2-time1);

    //Récupération des donnes du device vers le host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    //Désallocation du device memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    //Désallocation du host memory
    free(h_a); free(h_b); free(h_c);
    return 0;
}