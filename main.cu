
#include <hip/hip_runtime.h>
#include <stdio.h>

//les fonctions avec __global__ seront execute sur la CG (device)
__global__ void add(int* a, int* b, int* c){
    *c = *a + *b;
}

//Le reste est compile avec le compilateur de base genre gcc
int main(void) {
    int size = sizeof(int);

    //Variables présente sur le processeur (host)
    int h_a, h_b, h_c;

    //Pointeurs pour la device memory
    int *d_a, *d_b, *d_c;
    
    //Alloue la mémoire device
    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    //Valeurs de base
    h_a = 1687;
    h_b = 35148;

    //Copie les valeurs dans la device memory
    hipMemcpy(d_a, &h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, size, hipMemcpyHostToDevice);

    //Calcul de somme en parallèle
    add<<<1,1>>>(d_a, d_b, d_c);

    //Récupération des donnes du device vers le host
    hipMemcpy(&h_c, d_c, size, hipMemcpyDeviceToHost);

    //Désallocation de la device memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    printf("Calcul en parallele:\n%d + %d = %d\n", h_a, h_b, h_c);

    return 0;
}