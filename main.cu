#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/glut.h>

#include <cuda_gl_interop.h>

#include <unistd.h>
#include <stdio.h>
#include <time.h>

#define SCREEN_COL 1920
#define SCREEN_ROW 1080

#define TEXTUR_COL 4096
#define TEXTUR_ROW 2048

#define NB_THREAD 1024
#define VOISINAGE 3

#define DEVICE 0 //see the temp.cu for the device number

//Pour les boucles
bool state;
int my_window;
int size;
int cell_per_thread;

//Pointeurs pour la memoire
char *row1, *row2, *host_row;

//Pour l'affichage
GLuint gl_pixelBufferObject;
GLuint gl_texturePtr;
hipGraphicsResource* cudaPboResource;
uchar4* d_textureBufferData = nullptr;


//Epoque calcule sur le device. Un appel a cette fonction par pixel
__global__ void epoque(char* in, char* out, char*){
    uint x = threadIdx.x * (blockIdx.x*NB_THREAD) / 16;

    __shared__ char ligne[TEXTUR_COL];

    uint first=0;
    uint last=TEXTUR_COL-1;

    //Recopier la ligne pour les calculs.
    ligne[x] = in[x];

    //Attendre que le block ait finis de recopier les lignes
    __syncthreads();


}

//Calcule la texture à afficher
__global__ void affichageCuda(char* map, uchar4* texture){
    uint first_x = threadIdx.x;
    uint x;
    uint y = blockIdx.x;

    //Pour éviter de refaire les multiplication
    uint out_line = y * NB_COLONNE;
    uint pos;

    x = first_x;
    while(x < NB_COLONNE){
        pos = x + out_line;
        if(map[pos] == (char)1){
            texture[pos].x = 255;
            texture[pos].y = 255;
            texture[pos].z = 255;
        }
        else{
            texture[pos].x = 0;
            texture[pos].y = 0;
            texture[pos].z = 0;
        }

        x+=NB_THREAD;
    }
}

//Générer une carte de départ aléatoire
void random_map(char* map, int n){
    int i;

    for(i=0; i<n; i++){
        if(rand()%100 > 50)
            map[i] = 1;
        else
            map[i] = 0;
    }
}

void reset(){
    state = false;
    random_map(host_map,N);
    hipMemcpy(map1, host_map, size, hipMemcpyHostToDevice);
}

//Afficher la carte dans la console. Un # représente une case vivante, un < > une case morte.
//Un saut de ligne sépare chaque ligne. Si le lecteur (console) affiche automatiquement un saut de ligne cet affichage est inutile quand l'image est trop grande.
void affichageConsole(char* map){
    int i, j;
    for(j=0; j<NB_LIGNE; j++){
        for(i=0; i<NB_COLONNE; i++)
            printf((map[i + j*NB_COLONNE]?"#":" "));
        printf("\n");
    }
}

clock_t t_1 = clock();
//Mis à jours de la fenêtre.
//Déclenche des époques et le dessin de l'image.
void renderScene(void){
    static clock_t t_1 = clock();
    clock_t t, t_e, t_a;
    int k = 0;
    size_t num_bytes;

    //Temps pour les Epoques
    //t_e = clock();

    //Si FAST_SPEED est défini, on effectue un maximum d'époques entre deux frames, sinon une seul époque par frame
#ifdef FAST_SPEED
    do{
#endif
        state = !state;

        if(state)
            epoque<<<NB_LIGNE,NB_THREAD>>>(map1, map2);
        else
            epoque<<<NB_LIGNE,NB_THREAD>>>(map2, map1);
        
        k++;

#ifdef FAST_SPEED
        t = clock()-t_1;
    }while(t < max_time);
#endif
    //printf("  Epoques en %.5fs (%d)\n", (double)(clock()-t_e)/CLOCKS_PER_SEC, k);
    //Reset du timer ici. On prend en compte l'affichage pour le calcul du temps.
    t_1 = clock();

    //Temps pour l'Affichage
    //t_a = clock();

    //Affichage

    glClear(GL_COLOR_BUFFER_BIT);
    glEnable(GL_TEXTURE_2D);
    //Bind la texture
    glBindTexture(GL_TEXTURE_2D, gl_texturePtr);
    //Bind le PBO
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pixelBufferObject);

    //Calcul de l'image
    //On réserve le PBO
    hipGraphicsMapResources(1, &cudaPboResource, 0);
    hipGraphicsResourceGetMappedPointer((void**)&d_textureBufferData, &num_bytes, cudaPboResource);

    if(state)
        affichageCuda<<<NB_LIGNE,NB_THREAD>>>(map2, d_textureBufferData);
    else
        affichageCuda<<<NB_LIGNE,NB_THREAD>>>(map1, d_textureBufferData);

    //Copier les pixels du PBO
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, NB_COLONNE, NB_LIGNE, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    hipGraphicsUnmapResources(1, &cudaPboResource, 0);
   
    //On dessine la texture à l'écran
    glBegin(GL_QUADS);

    glTexCoord2f(0.0f, 0.0f);    glVertex2f(0.0f, 0.0f);
    glTexCoord2f(1.0f, 0.0f);    glVertex2f(float(TAILLE_LARGEUR), 0.0f);
    glTexCoord2f(1.0f, 1.0f);    glVertex2f(float(TAILLE_LARGEUR), float(TAILLE_HAUTEUR));
    glTexCoord2f(0.0f, 1.0f);    glVertex2f(0.0f, float(TAILLE_HAUTEUR));

    glEnd();
   
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    glBindTexture(GL_TEXTURE_2D, 0);

    glutSwapBuffers();
    //printf("Affichage en %.5fs\n", (double)(clock()-t_a)/CLOCKS_PER_SEC);
}

void exit_function(){
    printf("Exiting...\n");
    hipDeviceSynchronize();
    hipGraphicsUnregisterResource(cudaPboResource);

    hipFree(map1);
    hipFree(map2);

    free(host_map);

    exit(0);
}

//Gère les commandes clavier
void keyboardHandler(unsigned char key, int x, int y){
    //Permet de quitter le programme
    if(key==27){
        exit_function();
    }
    if(key=='r'){
        reset();
    }
}

bool initialisation_opengl(int& argc, char** argv){
    //init glut
    glutInit(&argc, argv);
    //Init windows
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(1920,1080);
    my_window = glutCreateWindow("Game Of Life");
    glutFullScreen();

    //event callbacks
    glutDisplayFunc(renderScene);
    glutIdleFunc(renderScene);
    glutKeyboardFunc(keyboardHandler);


    //Préparation de la texture
    
    glewInit();
    //Enable server side capabilities
    glEnable(GL_TEXTURE_2D);

    
    //On génère une texture dans le pointeur
    glGenTextures(1, &gl_texturePtr);
    //Bind le type de texture
    glBindTexture(GL_TEXTURE_2D, gl_texturePtr);
    //Quelques paramètres
        //Permet une texture cyclique
        //glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
        //glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
        
        //Si on zoom sur la texture, on utilise le nearest. (pas de flou, gros pixel)
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    //Défini la texture. Une GL_TEXTURE_2D, level de base, RGB avec Alpha sur 8bit, taille, pas de bord, pixel format rgba, pixel type, pointeur data
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, NB_COLONNE, NB_LIGNE, 0, GL_RGBA, GL_UNSIGNED_BYTE, 0);


    //Génère les buffers. Il y en as 1.
    glGenBuffers(1, &gl_pixelBufferObject);

    //Permet de bind le buffer et travailler dessus ensuite
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pixelBufferObject);

    //Créer et initialise le buffer. On copye h_textureBufferData dans le buffer d'openGL
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, NB_COLONNE * NB_LIGNE * sizeof(uchar4), 0, GL_STREAM_COPY);

    //Créer le Pixel Buffer Object. Cuda va écrire dedans, OpenGL va l'afficher. Rien ne passe par le CPU.
    hipError_t result = hipGraphicsGLRegisterBuffer(&cudaPboResource, gl_pixelBufferObject, cudaGraphicsMapFlagsWriteDiscard);
    if (result != hipSuccess) return false;

    //On un-bind tous les buffer & textures.
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    glBindTexture(GL_TEXTURE_2D, 0);

    //Change les coordonnees pour l'affichage
    glMatrixMode(GL_PROJECTION);
    glOrtho(0, NB_COLONNE, 0, NB_LIGNE, -1, 1);
    glMatrixMode(GL_MODELVIEW);

    return true;
}

//Le reste est compile avec le compilateur de base genre gcc
int main(int argc, char** argv) {

    srand (time (NULL));
    //Informations sur la map
    N = NB_COLONNE * NB_LIGNE;
    size = N * sizeof(char);

    //Variables de boucles
    max_time = CLOCKS_PER_SEC / FPS;
    state = false;

    //Alloue la mémoire device
    printf("Allocation Device\n");
    hipMalloc((void**) &map1, size);
    hipMalloc((void**) &map2, size);
    

    //Alloue la mémoire host
    printf("Allocation Host\n");
    host_map = (char*) malloc (size);
    
    //Attendre que la copie se termine
    hipDeviceSynchronize();

    //Désallocation du host memory

    //printf("Initialisation de la fenêtre\n");
    if(!initialisation_opengl(argc, argv))
        exit_function();

    //windows process
    printf("Execution\n");
    reset();
    glutMainLoop();
    
    //Pas de désallocation ici, le programme quitte dans le keyboard Handler.
    return 1;
}