#include "hip/hip_runtime.h"
#include "initialisation.cuh"
#include <boost/filesystem.hpp>

extern unsigned char *data1, *data2, *host_data;
extern unsigned long rule_id;
extern char* rule;
extern unsigned char voisinage, portee;
extern unsigned long texture_width, texture_height;
extern bool is_random;

void initialisation(int argc, char** argv){
    {boost::filesystem::path dir("./output/"); boost::filesystem::create_directory(dir);}
    {boost::filesystem::path dir("./output/random"); boost::filesystem::create_directory(dir);}
    {boost::filesystem::path dir("./output/random/3voisinage"); boost::filesystem::create_directory(dir);}
    {boost::filesystem::path dir("./output/random/5voisinage"); boost::filesystem::create_directory(dir);}
    {boost::filesystem::path dir("./output/random/7voisinage"); boost::filesystem::create_directory(dir);}
    {boost::filesystem::path dir("./output/one_seed"); boost::filesystem::create_directory(dir);}
    {boost::filesystem::path dir("./output/one_seed/3voisinage"); boost::filesystem::create_directory(dir);}
    {boost::filesystem::path dir("./output/one_seed/5voisinage"); boost::filesystem::create_directory(dir);}
    {boost::filesystem::path dir("./output/one_seed/7voisinage"); boost::filesystem::create_directory(dir);}

    texture_width = 3840;
    texture_height = 2160;
    
    portee = 1;
    voisinage = portee*2+1;
    uint size_data = texture_width;//*sizeof(char);
    

    //Alloue la mémoire device
    hipMalloc((void**) &data1, size_data);
    hipMalloc((void**) &data2, size_data);
    hipDeviceSynchronize();

    //Alloue la mémoire host
    host_data = (unsigned char*) malloc (size_data);

    rule = (char*) malloc(1024);
    rule_id = 0;
    is_random = 0;
    sprintf(rule, "%d", rule_id);

    initial_data();

    initialisation_opengl(argc, argv);
}

extern GLuint gl_pixelBufferObject;
extern GLuint gl_texturePtr;
extern hipGraphicsResource* cudaPboResource;

void initialisation_opengl(int argc, char** argv){
    //initialisation de glut
    glutInit(&argc, argv);

    
    //initialisation de la fenêtre
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE); //Mode RGB
    glutInitWindowSize(SCREEN_WIDTH,SCREEN_HEIGHT); //Taille de la fenêtre
    glutCreateWindow("Automate Cellulaire"); //Création de la fenêtre
    glutFullScreen(); //Plein écran


    //Callback
    glutDisplayFunc(renderScene); //Lors de l'affichage
    glutIdleFunc(renderScene); //Idle -> quand rien ne se passe
    glutKeyboardFunc(keyboardHandler); // évenements claviers

    //Préparation de la texture
    glewInit();//Initialisation dec glew
    glEnable(GL_TEXTURE_2D);//Activer les capacités server-side

    //Création de la texture
    glGenTextures(1, &gl_texturePtr); //initialisation
    glBindTexture(GL_TEXTURE_2D, gl_texturePtr); //Bind une texture pour travailler dessus
        
    //Paramètre: le zoom prend le NEAREST -> pas de flou lors du zoom, on aura des gros pixels
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    //Ne répète pas la texture.
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);

    //Défini la texture. Une GL_TEXTURE_2D, level de base, RGB avec Alpha sur 8bit, taille, pas de bord, pixel format rgba, pixel type, pointeur data
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, texture_width, texture_height, 0, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    //Génère les buffers. Il y en as 1.
    glGenBuffers(1, &gl_pixelBufferObject);

    //Permet de bind le buffer et travailler dessus ensuite
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pixelBufferObject);

    //Créer et initialise le buffer à 0
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, texture_width * texture_height * sizeof(uchar4), 0, GL_STREAM_COPY);

    //Créer le Pixel Buffer Object. Cuda va écrire dedans, OpenGL va l'afficher. Rien ne passe par le CPU.
    hipGraphicsGLRegisterBuffer(&cudaPboResource, gl_pixelBufferObject, cudaGraphicsMapFlagsWriteDiscard);

    //On un-bind tous les buffer & textures.
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    glBindTexture(GL_TEXTURE_2D, 0);

    //Changers les coodronées écrans en pixel
    //glMatrixMode(GL_TEXTURE);
    //glOrtho(-(TEXTUR_COL/2), TEXTUR_COL/2, -(TEXTUR_ROW/2), TEXTUR_ROW, -1, 1);
    //glMatrixMode(GL_PROJECTION);
    //glOrtho(0, SCREEN_WIDTH, 0, SCREEN_HEIGHT, -1, 1);
    //glMatrixMode(GL_MODELVIEW);
}