#include "hip/hip_runtime.h"
#include "affichage.cuh"



extern GLuint gl_pixelBufferObject;
extern GLuint gl_texturePtr;
extern hipGraphicsResource* cudaPboResource;
extern uchar4* d_textureBufferData;

extern unsigned char *data1, *data2, *host_data;
extern unsigned char *rule, *host_rule;

//Permet de calculer la proportion de l'image à afficher, car les coordonnées ne sont pas en pixel.
float l, r, u, d;
struct static_block{
    static_block(){
        //Affiche le centre horizontal de l'image
        l = (float(TEXTUR_COL)/2.0f - float(SCREEN_COL)/2.0f)/float(TEXTUR_COL);
        r = (float(TEXTUR_COL)/2.0f + float(SCREEN_COL)/2.0f)/float(TEXTUR_COL);
        //Affiche le haut de l'immage
        u = 0.0f;
        d = (float(SCREEN_ROW))/float(TEXTUR_ROW);
        //Affiche le bas de l'image
        //u = (float(TEXTUR_ROW) - float(SCREEN_ROW))/float(TEXTUR_ROW);
        //d = 1.0f;
    
        if(l < 0.0f) l = 0.0f;
        if(r > 1.0f) r = 1.0f;
        if(u < 0.0f) u = 0.0f;
        if(d > 1.0f) d = 1.0f;

        //Inverser le haut et le bas. Je ne sais pas pourquoi la texture est inversée
        float tmp;
        tmp = u;
        u = d;
        d = tmp;
    }
};
static static_block my_static_block;

//Fonction de boucle principale
void renderScene(void){
    uint i;
    bool state = 1;
    static size_t texture_size = TEXTUR_COL * TEXTUR_ROW * sizeof(uchar4);
    static uint NB_BLOCK = TEXTUR_COL / NB_THREAD;
    
    glClear(GL_COLOR_BUFFER_BIT); //Effacer l'écran
    glEnable(GL_TEXTURE_2D); //Active server-side
    glBindTexture(GL_TEXTURE_2D, gl_texturePtr); //Bind de la texture pour l'utiliser
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pixelBufferObject); //Bind du PBO pour l'utiliser
    hipGraphicsMapResources(1, &cudaPboResource, 0); //Bind de la texture cuda pour l'utiliser
    hipGraphicsResourceGetMappedPointer((void**)&d_textureBufferData, &texture_size, cudaPboResource); //Récupération du pointeur
    
    //Calcul de la 1e ligne de la texture
    texture_cuda<<<NB_BLOCK,NB_THREAD>>>(data1, d_textureBufferData, 0);
    hipMemcpy(host_data, data1, TEXTUR_COL, hipMemcpyDeviceToHost); hipDeviceSynchronize();
    print_data();

    //Boucle sur le reste des lignes de la texture
    for(i=1; i < TEXTUR_ROW; i++){
        if(state){
            //Effectuer une époque
            data_cuda<<<NB_BLOCK,NB_THREAD>>>(data1, data2, rule); hipDeviceSynchronize();
            //Calcul de la ligne de la texture
            texture_cuda<<<NB_BLOCK,NB_THREAD>>>(data2, d_textureBufferData, i); hipDeviceSynchronize();
            hipMemcpy(host_data, data2, TEXTUR_COL, hipMemcpyDeviceToHost);hipDeviceSynchronize();
        }
        else{
            data_cuda<<<NB_BLOCK,NB_THREAD>>>(data2, data1, rule); hipDeviceSynchronize();
            texture_cuda<<<NB_BLOCK,NB_THREAD>>>(data1, d_textureBufferData, i); hipDeviceSynchronize();
            hipMemcpy(host_data, data1, TEXTUR_COL, hipMemcpyDeviceToHost); hipDeviceSynchronize();
        }
        print_data();
        state = 1-state;
    }

    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, TEXTUR_COL, TEXTUR_ROW, GL_RGBA, GL_UNSIGNED_BYTE, 0); //Copier les pixels du PBO vers la texture gl
    hipGraphicsUnmapResources(1, &cudaPboResource, 0); //Désallouer la texture cuda
    
    //Afficher la texture à l'écran

    glBegin(GL_QUADS); //On dessine une texture dans un quadrilatère (rectangle de l'écran)
    

    //coordonnée texture (pixel)   -   coordonnées écran (pixel)
    glTexCoord2f( l, u);              glVertex2f(0.0f, 0.0f);
    glTexCoord2f( r, u);              glVertex2f(float(SCREEN_COL), 0.0f);
    glTexCoord2f( r, d);              glVertex2f(float(SCREEN_COL), float(SCREEN_ROW));
    glTexCoord2f( l, d);              glVertex2f(0.0f, float(SCREEN_ROW));

    glEnd(); //Fin du quadrilatère
   
    //Libérer les buffer
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    glBindTexture(GL_TEXTURE_2D, 0);

    glutSwapBuffers();
    initial_data();
}

//Affichage d'une ligne dans le terminal
void print_data(){
    uint i;
    
    for(i=0; i<TEXTUR_COL; i++)
        printf("%c",(host_data[i]?'#':' '));

    printf("\n");
}


extern uint rule_id;
//Affichage de la règle dans le terminal
void print_rule(){
    printf("Rule number: %d\n", rule_id);
    uint i;
    uint nb_state = pow(2, VOISINAGE*2+1);

    printf(":   :  #: # : ##:#  :# #:## :###:\n");

    for(i=0; i<nb_state; i++)
        printf("%s", (host_rule[i]==1?": # ":":   "));
    
    printf(":\n\n");
}